#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <cnmem.h>
#include <stdint.h>
#include <fstream>

static std::size_t getFreeMemory() {
    hipFree(0);
    std::size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    return freeMem;
}

class CnmemTest : public ::testing::Test {
    /// We determine the amount of free memory.
    std::size_t mFreeMem;
    
protected:
    /// Do we test memory leaks.
    bool mTestLeaks;
    /// Do we skip finalization.
    bool mFinalize;
    
public:
    /// Ctor.
    CnmemTest() : mFreeMem(getFreeMemory()), mTestLeaks(true), mFinalize(true) {}
    /// Tear down the test.
    void TearDown();
};

void CnmemTest::TearDown() {
    if( mFinalize ) {
        ASSERT_EQ(CNMEM_STATUS_SUCCESS, cnmemFinalize()); 
    }
    if( mTestLeaks ) {
        ASSERT_EQ(mFreeMem, getFreeMemory());
    }
    hipDeviceReset();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<typename T>
__global__ void tinyKernel(T* d_a, int numElem)
{
    int ind = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(ind >= numElem)
        return;
    d_a[ind] += 1;
}


struct _24ByteStruct
{
    double a;
    double c;
    double b;

    __host__ __device__
    void operator +=(int other)
    {
        a += other;
        b += other;
        c += other;
    }

    __host__ __device__
    void operator =(int other)
    {
        a = other;
        b = other;
        c = other;
    }
};

template<typename T, int expectedSize>
void testAlign()
{
    const int numElem = 200;
    const int size = numElem*sizeof(T);
    T* cpuData = new T[numElem];
    for(int i = 0; i < numElem; i++)
        cpuData[i] = i;

    ASSERT_EQ(expectedSize, sizeof(T));

    hipStream_t streams[2];
    ASSERT_EQ(hipSuccess, hipStreamCreate(&streams[0]));
    ASSERT_EQ(hipSuccess, hipStreamCreate(&streams[1]));

    cnmemDevice_t device;
    memset(&device, 0, sizeof(device));
    device.numStreams = 2;
    device.streams = streams;
    //intentonally misallign, but could be from calculation based on gpu size
    size_t streamSizes[] = { size*2 + sizeof(T) - 1, size*2 + sizeof(T) - 1 };
    device.streamSizes = streamSizes;

    ASSERT_EQ(CNMEM_STATUS_SUCCESS, cnmemInit(1, &device, CNMEM_FLAGS_DEFAULT));
    T *ptr0, *ptr1;
    ASSERT_EQ(CNMEM_STATUS_SUCCESS, cnmemMalloc((void**)&ptr0, size, streams[0]));
    ASSERT_EQ(CNMEM_STATUS_SUCCESS, cnmemMalloc((void**)&ptr1, size, streams[1]));

    ASSERT_EQ(hipSuccess, hipMemcpyAsync(ptr0, cpuData, size, hipMemcpyHostToDevice, streams[0]));
    ASSERT_EQ(hipSuccess, hipMemcpyAsync(ptr1, cpuData, size, hipMemcpyHostToDevice, streams[1]));

    //force read and write from ptr0,1
    tinyKernel<<<numElem, 1, 0, streams[0]>>>(ptr0, numElem);
    tinyKernel<<<numElem, 1, 0, streams[1]>>>(ptr1, numElem);

    ASSERT_EQ(hipSuccess, hipStreamSynchronize(streams[0]));
    ASSERT_EQ(hipSuccess, hipStreamSynchronize(streams[1]));

    ASSERT_EQ(CNMEM_STATUS_SUCCESS, cnmemFree(ptr1, streams[1])); 
    ASSERT_EQ(CNMEM_STATUS_SUCCESS, cnmemFree(ptr0, streams[0]));
    
    ASSERT_EQ(hipSuccess, hipStreamDestroy(streams[0]));
    ASSERT_EQ(hipSuccess, hipStreamDestroy(streams[1]));

    ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
}

TEST_F(CnmemTest, alignment8) {
    testAlign<char, 1>();
}

TEST_F(CnmemTest, alignment16) {
    testAlign<short, 2>();
}

TEST_F(CnmemTest, alignment32) {
    testAlign<float, 4>();
}

TEST_F(CnmemTest, alignment64) {
    testAlign<double, 8>();
}

TEST_F(CnmemTest, alignment192) {
    testAlign<_24ByteStruct, 24>();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}

///////////////////////////////////////////////////////////////////////////////////////////////////

